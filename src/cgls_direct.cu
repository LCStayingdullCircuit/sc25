#include "hip/hip_runtime.h"
#include "LATER.h"
#include "utils.h"
#include "matrix_generation.h"
#include "qr_decomposition.h"
#include <random> 

using namespace Utils;
#define threadsPerDim 16

typedef int INT;
long int m, n, nb, b;
int itera;
long int m1, n1;
int datatype, condition;
#ifndef CGLS_DISABLE_ERROR_CHECK
#define CGLS_CUDA_CHECK_ERR()                                                   \
    do                                                                          \
    {                                                                           \
        hipError_t err = hipGetLastError();                                   \
        if (err != hipSuccess)                                                 \
        {                                                                       \
            printf("%s:%d:%s\n ERROR_CUDA: %s\n", __FILE__, __LINE__, __func__, \
                   hipGetErrorString(err));                                    \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    } while (0)
#else
#define CGLS_CUDA_CHECK_ERR()
#endif


// 确认输入参数，不涉及数据类型
int parseArguments(int argc, char *argv[])
{
    if (argc < 7)
    {
        printf("Needs m, n and nb as inputs\n");
        return -1;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    nb = atoi(argv[3]);
    datatype = atoi(argv[4]);
    condition = atoi(argv[5]);
    itera = atoi(argv[6]);
    return 0;
}

// Numeric limit epsilon for float, double, complex_float, and complex_double.
template <typename T>
double Epsilon();

template <>
inline double Epsilon<double>()
{
    return std::numeric_limits<double>::epsilon();
}

template <>
inline double Epsilon<hipDoubleComplex>()
{
    return std::numeric_limits<double>::epsilon();
}

template <>
inline double Epsilon<float>()
{
    return std::numeric_limits<float>::epsilon();
}

template <>
inline double Epsilon<hipFloatComplex>()
{
    return std::numeric_limits<float>::epsilon();
}

template <typename T>  
int Solve(hipblasHandle_t handle,  
          const T* A,  
          const int m,  
          const int n,  
          const T* b,  
          T* x,  
          const double shift,  
          const double tol,  
          const int maxit,  
          bool quiet,  
          std::vector<double>& normsRatios)  
{  

    T *p = nullptr, *q = nullptr, *r = nullptr, *s = nullptr;  
    double gamma = 0.0, normp = 0.0, normq = 0.0, norms = 0.0, norms0 = 0.0;  
    double normx = 0.0, xmax = 0.0;  
    int k = 0, flag = -1, indefinite = 0;  
    const double kEps = Epsilon<T>();  
    const T kOne = static_cast<T>(1.0);  
    const T kZero = static_cast<T>(0.0);  
    const T kNeg1 = static_cast<T>(-1.0);  

    hipMalloc(&p, n * sizeof(T));  
    hipMalloc(&q, m * sizeof(T));  
    hipMalloc(&r, m * sizeof(T));  
    hipMalloc(&s, n * sizeof(T));  

    hipMemcpy(r, b, m * sizeof(T), hipMemcpyDeviceToDevice);  
    hipMemcpy(s, x, n * sizeof(T), hipMemcpyDeviceToDevice);  

    nrm2(handle, n, x, &normx);  
    hipDeviceSynchronize();  

    if (normx > 0.0)  
    {  
        double alpha = -1.0, beta = 1.0;  
        hipblasDgemv(handle, HIPBLAS_OP_N,  
                    m, n,  
                    &alpha, (const double*)A, m,  
                    (const double*)x, 1,  
                    &beta, (double*)r, 1);  
    }  

    {  
        double alpha = 1.0, zero = 0.0;  
        hipblasDgemv(handle, HIPBLAS_OP_T,  
                    m, n,  
                    &alpha,  
                    (const double*)A, m,  
                    (const double*)r, 1,  
                    &zero,  
                    (double*)s, 1);  
    }  
    hipDeviceSynchronize();  

    hipMemcpy(p, s, n * sizeof(T), hipMemcpyDeviceToDevice);  
    nrm2(handle, n, s, &norms);  
    nrm2(handle, n, x, &normx);  
    hipDeviceSynchronize();  

    norms0 = norms;  
    gamma  = norms * norms;  
    xmax   = normx;  


    for (k = 0; k < maxit; ++k)  
    {  
        double alpha = 1.0, zero = 0.0;  
        hipblasDgemv(handle, HIPBLAS_OP_N,  
                    m, n,  
                    &alpha,  
                    (const double*)A, m,  
                    (const double*)p, 1,  
                    &zero,  
                    (double*)q, 1);  

        hipDeviceSynchronize();  
        nrm2(handle, n, p, &normp);  
        nrm2(handle, m, q, &normq);  

        double delta = normq * normq + shift * normp * normp;  
        if (delta <= 0.0) indefinite = 1;  
        if (delta == 0.0) delta = kEps;  

        double alphaD = gamma / delta;  
        double negAlphaD = -(gamma / delta);  

        axpy(handle, n, &alphaD, p, 1, x, 1);  
        axpy(handle, m, &negAlphaD, q, 1, r, 1);  

        {  
            double alpha2 = 1.0, zero2 = 0.0;  
            hipblasDgemv(handle, HIPBLAS_OP_T,  
                        m, n,  
                        &alpha2,  
                        (const double*)A, m,  
                        (const double*)r, 1,  
                        &zero2,  
                        (double*)s, 1);  
        }  

        hipDeviceSynchronize();  
        nrm2(handle, n, s, &norms);  

        double gammaOld = gamma;  
        gamma = norms * norms;  
        double betaD = gamma / gammaOld;  

        axpy(handle, n, &betaD, p, 1, s, 1);  
        hipMemcpy(p, s, n * sizeof(T), hipMemcpyDeviceToDevice);  

        nrm2(handle, n, x, &normx);  
        hipDeviceSynchronize();  

        xmax = std::max(xmax, normx);  
        bool converged = (norms <= norms0 * tol) || (normx * tol >= 1.0);  
        normsRatios.push_back(norms / norms0);  
        if (converged)  
        {  
            // flag = 1;
            break;  
        }  

    }  

    if(k < maxit - 1) {  
        flag = 1;  
    }  
    else {  
        flag = -1;  
    }  
    printf("iteration times is %d\n", k);
    if(itera == 5) {
        std::string text = ".txt";  
        std::string resultString = "res" + std::to_string(datatype)+ "_" + std::to_string(condition)+ "_nonpre" + text;  
        std::ofstream outFile(resultString);  
        for (double res : normsRatios) {  
            outFile << res << std::endl;  
        }  
    }
    hipFree(p);  
    hipFree(q);  
    hipFree(r);  
    hipFree(s);  
    return flag;  
}  


int main(int argc, char *argv[])  
{  
    if (parseArguments(argc, argv) == -1)  
        return 0;  
    double *dtA;   
    hipMalloc(&dtA, sizeof(double) * m * n);  
    generateMatrix(dtA, m, n, condition, datatype);  

    // float *A;           hipMalloc(&A, sizeof(float) * m * n);  
    // float *R;           hipMalloc(&R, sizeof(float) * n * n);  
    // float *work;        hipMalloc(&work, sizeof(float) * m * n);  
    // __half *halfwork;   hipMalloc(&halfwork, sizeof(__half) * m * n);  
    // double *dR;         hipMalloc(&dR, sizeof(double) * n * n);  
    
    hipsolverHandle_t cusolver_handle;  
    hipblasHandle_t     cublas_handle;  
    cudaCtxt           ctxt;  
    hipsolverDnCreate(&cusolver_handle);  
    hipblasCreate(&cublas_handle);  
    hipblasCreate(&ctxt.cublas_handle);  

    std::vector<double> b(m, 1.0);  
    std::vector<double> x(n, 0.0);  
    double *d_b, *d_x;  
    hipMalloc(&d_b, sizeof(double) * m);  
    hipMalloc(&d_x, sizeof(double) * n);  
    hipMemcpy(d_b, b.data(), sizeof(double) * m, hipMemcpyHostToDevice);  
    hipMemcpy(d_x, x.data(), sizeof(double) * n, hipMemcpyHostToDevice);  
    double shift = 0.0;  
    double tol = 1e-13;  
    bool quiet = false;  
    int Iternumber = 0;  
    std::vector<double> normsRatios;  
    mystartTimer();
    int flag = Solve<double>(cublas_handle, dtA, m, n, d_b, d_x, shift, tol, itera, quiet, normsRatios);  
    float time = mystopTimer();
    printf("direct iteration takes %.4f ms\n", time);
    // hipFree(work);  
    // hipFree(halfwork);   
    hipFree(dtA);  
    // hipFree(A);  
    // hipFree(R);  
    hipFree(d_b);  
    hipFree(d_x);  
    // hipFree(dR);  

    hipblasDestroy(cublas_handle);  
    hipsolverDnDestroy(cusolver_handle);  
    hipblasDestroy(ctxt.cublas_handle);  

    return 0;  
}  