#include "LATER.h"
#include "utils.h"
#include "matrix_generation.h"
#include "qr_decomposition.h"
#include <random> 

using namespace Utils;
#define threadsPerDim 16

typedef int INT;
long int m, n, nb, b;
long int m1, n1;
int datatype, condition;

int parseArguments(int argc, char *argv[])
{
    if (argc < 7)
    {
        printf("Needs m, n and nb as inputs\n");
        return -1;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    nb = atoi(argv[3]);
    b = atoi(argv[4]);
    datatype = atoi(argv[5]);
    condition = atoi(argv[6]);
    return 0;
}

void generateQ(  
    int m, int n, int nb,  
    double* d_W,       
    double* d_Y,        
    double* d_Q,              
    double* work,                      
    hipblasHandle_t cublas_handle)  
{  
    int num_blocks = n / nb;  
    double alpha = 1.0;  
    double beta = 0.0;  
    double negalpha = -1.0;
    
    for (int k = num_blocks - 1; k >= 0; k--) {  
        int start_row = k * nb;  
        int current_m = m - start_row;
          
        double* W_k = d_W + start_row * m + start_row; 
        double* Y_k = d_Y + start_row * m + start_row; 
        double* Q_k = d_Q + start_row * m + start_row;

        hipblasDgemm(cublas_handle,  
                    HIPBLAS_OP_T, HIPBLAS_OP_N,  
                    nb, current_m, current_m,  
                    &alpha, Y_k, m,  
                    Q_k, m,          
                    &beta, work, nb); 

        hipblasDgemm(cublas_handle,  
                    HIPBLAS_OP_N, HIPBLAS_OP_N,  
                    current_m, current_m, nb,  
                    &negalpha, W_k, m,     
                    work, nb,              
                    &alpha, Q_k, m);       
    }  
}  
void SolveWithIterativeQR_Double(  
    int m, int n, int nb,  
    const double* d_W,  
    const double* d_Y,  
    const double* d_R,  
    double* d_b,  
    double* d_x,  
    hipblasHandle_t cublas_handle)  
{  
    double* d_temp = nullptr;  
    hipMalloc(&d_temp, nb * sizeof(double));  

    const double alpha = 1.0;  
    const double beta  = 0.0;  
    const double neg_one = -1.0;  

    for (int i = 0; i < n; i += nb)  
    {  
        int kb = ((i + nb) <= n) ? nb : (n - i);  

        const double* d_Wi = d_W + i * m;  
        const double* d_Yi = d_Y + i * m;  

        hipblasDgemv(  
            cublas_handle,  
            HIPBLAS_OP_T,  
            m, kb,  
            &alpha,  
            d_Wi, m,  
            d_b, 1,  
            &beta,  
            d_temp, 1);  

        hipblasDgemv(  
            cublas_handle,  
            HIPBLAS_OP_N,  
            m, kb,  
            &neg_one,  
            d_Yi, m,  
            d_temp, 1,  
            &alpha,  
            d_b, 1);  
    }  

    hipblasDcopy(cublas_handle, n, d_b, 1, d_x, 1);  

    hipblasDtrsv(  
        cublas_handle,  
        HIPBLAS_FILL_MODE_UPPER,  
        HIPBLAS_OP_N,  
        HIPBLAS_DIAG_NON_UNIT,  
        n,  
        d_R, n,  
        d_x, 1);  

    hipFree(d_temp);  
}  

int main(int argc, char *argv[])  
{  
    if (parseArguments(argc, argv) == -1)  
        return 0;  

    int lda = m;  
    const int ldwork1 = m + 108 * nb / 2;  
    double *dtA;   hipMalloc(&dtA, sizeof(double) * m * n);  
    generateMatrix(dtA, m, n, condition, datatype);  

    double *W;     hipMalloc(&W, sizeof(double) * m * n);  
    double *d_Y;   CHECK_CUDA(hipMalloc(&d_Y, sizeof(double) * m * n));  
    double *d_Q;   CHECK_CUDA(hipMalloc(&d_Q, sizeof(double) * m * m));  
    double *R;     hipMalloc(&R, sizeof(double) * n * n);  
    double *work1; CHECK_CUDA(hipMalloc(&work1, ldwork1 * 1024 * sizeof(double)));  
    double *work2; CHECK_CUDA(hipMalloc(&work2, m * n * sizeof(double)));  
    double *work3; CHECK_CUDA(hipMalloc(&work3, m * nb * sizeof(double)));  
    CHECK_CUDA(hipMemset(W,   0, m * n * sizeof(double)));  
    CHECK_CUDA(hipMemset(d_Y, 0, m * n * sizeof(double)));  

    std::vector<double> hb(m, 1.0);  
    std::vector<double> hx(n, 0.0);  
    double *d_b, *d_x;  
    hipMalloc(&d_b, sizeof(double) * m);  
    hipMalloc(&d_x, sizeof(double) * n);  
    hipMemcpy(d_b, hb.data(), sizeof(double) * m, hipMemcpyHostToDevice);  
    hipMemcpy(d_x, hx.data(), sizeof(double) * n, hipMemcpyHostToDevice);  

    hipsolverHandle_t cusolver_handle;  
    hipblasHandle_t     cublas_handle;  
    cudaCtxt           ctxt;  

    hipsolverDnCreate(&cusolver_handle);  
    hipblasCreate(&cublas_handle);  
    hipblasCreate(&ctxt.cublas_handle);  

    dim3 blockDim(threadsPerDim, threadsPerDim);  
    dim3 gridDim((m + threadsPerDim - 1) / threadsPerDim,  
                 (m + threadsPerDim - 1) / threadsPerDim);  

    mystartTimer();  

    IterativeQR(dtA, m, n, nb, b,  W, d_Y, lda, d_Q, R,  
                work1, work2, work3, ldwork1, m*n, cublas_handle);  

    setEye<<<gridDim, blockDim>>>(d_Q, m);  

    generateQ(m, n, nb, W, d_Y, d_Q, work2, ctxt.cublas_handle);  
    float highQRTime = mystopTimer();

    mystartTimer();
    SolveWithIterativeQR_Double(m, n, nb, W, d_Y, dtA, d_b, d_x, cublas_handle); 
    hipDeviceSynchronize();  
    float highComputeTime = mystopTimer();


    printf("high precision QR takes %.4f ms, high precision computation takes %.4f ms\n", highQRTime, highComputeTime);  

    hipFree(dtA);  
    hipFree(W);  
    hipFree(d_Y);  
    hipFree(d_Q);  
    hipFree(R);  
    hipFree(work1);  
    hipFree(work2);  
    hipFree(work3);  
    hipFree(d_x);
    hipFree(d_b);
    hipblasDestroy(cublas_handle);  
    hipsolverDnDestroy(cusolver_handle);  
    hipblasDestroy(ctxt.cublas_handle);  

    return 0;  
}  